#include "hip/hip_runtime.h"
// CUDA kernel -- Generate dot product

#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include "error_util.h"

using namespace std;
const int THREADS_NUM = 1024;

__global__ static void dot_product(float * a, float * b, float * c, int array_size)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
     
    if (tid < array_size)
    {
        extern __shared__ float c_tmp[];

        c_tmp[tid] = a[tid] * b[tid];
        __syncthreads();

        size_t length = array_size;
        size_t offset = (length - 1)/2 + 1;

        while (length >= 2)
        {
            if (tid + offset < length)
            {
                c_tmp[tid] = c_tmp[tid] + c_tmp[tid + offset];
                //__syncthreads();
            }

            length = (length - 1)/2 + 1;
            offset = (offset - 1)/2 + 1;
            __syncthreads(); 
        }

        c[0] = c_tmp[0];

//        c[tid] = a[tid] + b[tid];
    }
}

int main()
{
    int length = THREADS_NUM;
    float * a = new float [length];
    float * b = new float [length];
    float c = 0;

    for (size_t i = 0; i < length; i++)
    {
        a[i] = 1.5;
        b[i] = 2.5;
    }

    hipSetDevice(1);

    // Create CUDA event
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    checkCudaErrors(hipEventRecord(start, 0));
    // prepare data on GPU
    float * gpu_a;
    float * gpu_b;
    float * gpu_c;
    checkCudaErrors(hipMalloc((void **)&gpu_a, sizeof(float) * length));
    checkCudaErrors(hipMalloc((void **)&gpu_b, sizeof(float) * length));
    checkCudaErrors(hipMalloc((void **)&gpu_c, sizeof(float) * length));
    checkCudaErrors(hipMemcpy(gpu_a, a, sizeof(float) * length, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(gpu_b, b, sizeof(float) * length, hipMemcpyHostToDevice));

    // kernel function
    size_t blockNum = 1;
    size_t threadsNum = THREADS_NUM;
    dot_product<<<blockNum, threadsNum, sizeof(float) * length>>>(gpu_a, gpu_b, gpu_c, length);
    
    // Copy data back
    hipMemcpy(&c, gpu_c, sizeof(float), hipMemcpyDeviceToHost);

    // output
    cout<<"Sum = "<<c<<endl;

    checkCudaErrors(hipFree(gpu_a));
    checkCudaErrors(hipFree(gpu_b));
    checkCudaErrors(hipFree(gpu_c));
    delete [] a;
    delete [] b;

    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize( stop ));
    float elapsedTime;
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
    printf( "Time to generate dot product: %.3f ms\n", elapsedTime );

    return 0;
}
